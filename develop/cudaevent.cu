
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main() {
	hipEvent_t start_d;
	hipEvent_t stop_d;
	hipEventCreate(&start_d);
	hipEventCreate(&stop_d);

	hipEventRecord(start_d);
	hipEventRecord(stop_d);
	while (hipEventQuery(stop_d) != hipSuccess) {
		printf("not finished\n");
	}
	hipEventSynchronize(stop_d);

	float ms;
	hipEventElapsedTime(&ms, start_d, stop_d);
	printf("%f\n", ms);

	hipEventDestroy(start_d);
	hipEventDestroy(stop_d);

	return 0;
}