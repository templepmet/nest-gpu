#include <cstdio>
#include <mpi.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv)
{
    int rank;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int n = 4;
    const int p = 2;
    int *d, *h;
    MPI_Status st;
    MPI_Request req;

    hipMalloc(&d, sizeof(int) * n);
    hipHostMalloc(&h, sizeof(int) * n);

    if (rank == 1) {
       for (int i = 0; i < n; ++i) {
            h[i] = i + 1;
        }
        hipMemcpy(d, h, sizeof(int) * n, hipMemcpyHostToDevice);
    }

    if (rank == 0) {
        for (int i = 0; i < n; ++i) {
            printf("%d ", h[i]);
        }
        puts("");
    }

    if (rank == 1) {
        MPI_Isend(d, n, MPI_INT, 0, 0, MPI_COMM_WORLD, &req);
        MPI_Waitall(1, &req, &st);
    }
    else if (rank == 0) {
        MPI_Irecv(d, n, MPI_INT, 1, 0, MPI_COMM_WORLD, &req);
        MPI_Waitall(1, &req, &st);
    }

    hipMemcpy(h, d, sizeof(int) * n, hipMemcpyDeviceToHost);

    if (rank == 0) {
        for (int i = 0; i < n; ++i) {
            printf("%d ", h[i]);
        }
        puts("");
    }

    hipFree(d);
    hipHostFree(h);
    MPI_Finalize();

    return 0;
}
